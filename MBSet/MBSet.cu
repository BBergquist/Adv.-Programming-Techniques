#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"

#include <GL/freeglut.h>

// Size of window in pixels, both width and height
#define WINDOW_DIM            512

using namespace std;

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;
char* pixelBuf = new char[WINDOW_DIM * WINDOW_DIM * 3];
const int maxIt = 2000; // Maximum Iterations

// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values

__global__ void devDrawPix(float* realRange, float* imagRange, Complex* min, int* pixel)
{
  int threadId = threadIdx.x + blockIdx.x * blockDim.x;
  int x = threadId % WINDOW_DIM;
  int y = threadId / WINDOW_DIM;

  Complex cur(min->r + (x * *realRange)/WINDOW_DIM,
              min->i + (y * *imagRange)/WINDOW_DIM);
  Complex z(cur);
  for (pixel[threadId] = 0; pixel[threadId] < 2000; pixel[threadId]++)
  {
    z = z*z + cur;
    if (z.magnitude2() > 4.0)
    {
      break;
    }
  }
}

void display(void)
{
  float realRange, imagRange; // Host copies
  int* iters; // Host copy
  float* d_realRange,* d_imagRange; // Device copies
  int* d_iters; // Devide copy
  int size = WINDOW_DIM*WINDOW_DIM*sizeof(int);
  // Allocate memory for host
  realRange = maxC.r - minC.r;
  imagRange = maxC.i - minC.i;
  iters = (int*) malloc(size);
  // Allocate memory for device
  hipMalloc((void **)&d_realRange, sizeof(float));
  hipMalloc((void **)&d_imagRange, sizeof(float));
  hipMalloc((void **)&d_iters, size);
  hipMalloc((void **)&dev_minC, sizeof(Complex));
  // Copy inputs to device
  hipMemcpy(d_realRange, &realRange, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_imagRange, &imagRange, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_minC, &minC, sizeof(Complex), hipMemcpyHostToDevice);

  // Calculate pixels...
  devDrawPix<<<WINDOW_DIM*WINDOW_DIM/32,32>>>(d_realRange, d_imagRange, dev_minC, d_iters);

  // Copy results
  hipMemcpy(iters, d_iters, size, hipMemcpyDeviceToHost);

  // Populate pixelBuf based off of iterations
  for (int i = 0; i < WINDOW_DIM*WINDOW_DIM; i++)
  {
    pixelBuf[i*3] = (char) (colors[iters[i]].r * 255);
    pixelBuf[i*3 + 1] = (char) (colors[iters[i]].g * 255);
    pixelBuf[i*3 + 2] = (char) (colors[iters[i]].b * 255);
  }

  // Cleanup
  hipFree(d_realRange); hipFree(d_imagRange); hipFree(d_iters); hipFree(dev_minC);
  free(iters);
  
  glDrawPixels(WINDOW_DIM,WINDOW_DIM,GL_RGB,GL_UNSIGNED_BYTE,pixelBuf);
  glutSwapBuffers();
  glutPostRedisplay();
}

void init()
{
  glClearColor(0,0,0,0);
  glShadeModel(GL_FLAT);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  int w = glutGet(GLUT_WINDOW_WIDTH);
  int h = glutGet(GLUT_WINDOW_HEIGHT);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
  glOrtho(0.0, 1.0, 0.0, 1.0, -1.0, 1.0);
  glClear(GL_COLOR_BUFFER_BIT);
}

//void reshape(int w, int h)
//{
//  glViewport(0,0,(GLsizei)w,(GLsizei)h);
//  glMatrixMode(GL_PROJECTION);
//  glLoadIdentity();
//  glOrtho(0.0,(GLdouble)w,(GLdouble)0.0,h,(GLdouble)w*2,(GLdouble)-w*2);
//  glMatrixMode(GL_MODELVIEW);
//  glLoadIdentity();
//}

void InitializeColors()
{
  colors = new RGB[maxIt + 1];
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 5)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          srand48(333333333333333333 * drand48());
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); // black
}

int main(int argc, char** argv)
{
  // Initialize OPENGL here
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(WINDOW_DIM, WINDOW_DIM);
  glutInitWindowPosition(50, 50);
  glutCreateWindow("Mandelbrot Set");
  init();
  // Set up necessary host and device buffers
  // set up the opengl callbacks for display, mouse and keyboard
  glutDisplayFunc(display);
//  glutReshapeFunc(reshape);
  // Calculate the interation counts
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels
  InitializeColors();
  glutMainLoop(); // THis will callback the display, keyboard and mouse
  return 0;
  
}
